#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

__constant__ double W_y = 2;
__constant__ double W_max = 2;
__constant__ double R = 6;
__constant__ double PI = 3.14159265;

/*
struct vector {
    double x, y, z;
};
*/


typedef struct vectData {
	double x;
	double y;
	double z;
} vector;

__device__ vector create_vector(double x, double y, double z)
{
	vector vec;
	vec.x = x;
	vec.y = y;
	vec.z = z;
	return vec;
}



__global__ void initCurandStates(hiprandState *states, int numStates) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < numStates) {
        // Use id * 4238811 as the seed for each state for uniqueness
        hiprand_init(id * 4238811ULL, 0, 0, &states[id]);
    }
}

void write_to_file(float *G, int n, char *file_name) {
    FILE *fp = fopen(file_name, "wb"); // Open file in binary write mode
    if (fp == NULL) {
        perror("Error opening file");
        return;
    }

    for (int i = (n-1); i >= 0; i--) {
        // Write a row of n floats to the file as binary data
        if (fwrite(&G[i*n], sizeof(float), n, fp) != n) {
            perror("Error writing to file");
            break;
        }
    }
    fclose(fp);
}

__global__ void ray_tracing(float *G, hiprandState *states, int *samples, int N_rays, int n, int total_threads, double L_x, double L_y, double L_z, double C_y) {
    
    //printf("total_threads:%d\n", total_threads);
    //printf("N_rays:%d\n", total_threads);
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState localstate = states[id];
    
    //int stride = blockDim.x * gridDim.x;   //total threads (nblocks*ntpb)

    /* Predefined */
    vector L = create_vector(L_x, L_y, L_z);	
    vector C = create_vector(0, C_y, 0);	
    vector W = create_vector(0, W_y, 0);	

    //vector W = {0, W_y, 0};
    //vector C = {0, C_y, 0};
    //vector L = {L_x, L_y, L_z};

    //printf("size:%d\n",W.y);

    vector V,I,N,S;

    //printf("N_rays:%d\n", N_rays);
    //printf("total_threads:%d\n", total_threads);
        
    for(int ray = id; ray < N_rays; ray += total_threads){
        

        double temp = -1;
        int s = 0;
        
        while ((fabs(W.x) > W_max) || (fabs(W.z) > W_max) || temp <= 0.0) {
        //while ((W.x > W_max) || (W.x < -1*W_max) || (W.z > W_max) || (W.z < -1*W_max)|| temp <= 0) {
            
            double phi = 1 * PI * hiprand_uniform(&localstate);
            double cosTheta = 2 * hiprand_uniform(&localstate) - 1;
            double sinTheta = sqrt(1 - (cosTheta * cosTheta));

            V.x = sinTheta * (double) cos(phi);
            V.y = sinTheta * (double) sin(phi);
            V.z = cosTheta;

            s+=1;

            if (V.y != 0) {
                W.x = (W.y / V.y) * V.x;
                W.z = (W.y / V.y) * V.z;

                double vc = (V.x * C.x) + (V.y * C.y) + (V.z * C.z);
                double cc = (C.x * C.x) + (C.y * C.y) + (C.z * C.z);
                temp = (vc * vc) + (R * R) - cc;
            }
        }

        //printf("W.x %f,W.z%f,temp%f\n",W.x,W.z,temp );

        double t = (V.x * C.x) + (V.y * C.y) + (V.z * C.z) - sqrt(temp);
        I.x = t * V.x;
        I.y = t * V.y;
        I.z = t * V.z;

        N.x = (I.x - C.x) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));
        N.y = (I.y - C.y) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));
        N.z = (I.z - C.z) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));

        S.x = (L.x - I.x) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));
        S.y = (L.y - I.y) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));
        S.z = (L.z - I.z) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));

        double b = (S.x * N.x) + (S.y * N.y) + (S.z * N.z);
        if (b<0)
            b = 0;

        double normalizedWx = (W.x + W_max) / (2 * W_max);
        int i_index = (int)(normalizedWx * n);
        if (i_index < 0) i_index = 0;
        if (i_index >= n) i_index = n - 1;

        double normalizedWz = (W.z + W_max) / (2 * W_max);
        int j_index = (int)(normalizedWz * n);
        if (j_index < 0) j_index = 0;
        if (j_index >= n) j_index = n - 1;

        assert( (i_index < n) && (i_index>=0) );
        assert( (j_index < n) && (j_index>=0) );

        //printf("B:%f\n", (float) b);

        atomicAdd(&G[j_index + (n * i_index)], (float) b);
        //printf("s:%d\n",s);
        *samples+=s;
        //printf("G: %f\n", G[j_index + (n * i_index)]);
    }
    // bug fix, set the state again to ensure continuity 
    states[id] = localstate;
}


int main(int argc, char * argv[]) {

    if (argc != 5)
	{
		printf("4 arguments needed: number of rays, number of grid points, number of blocks, and threads per block. Exiting.\n");
		return 0;
	}

    clock_t start_total, end_total;
    start_total = clock();

    int N_rays = atoi(argv[1]);
    int n = atoi(argv[2]);
    int nblocks = atoi(argv[3]);
    int ntpb = atoi(argv[4]);

    int samples = 0;
    int *d_samples;

    hipMalloc( (void **) &d_samples, sizeof(int));
    hipMemcpy(d_samples, &samples, sizeof(int), hipMemcpyHostToDevice);


    int total_threads = nblocks * ntpb;
    
    double C_y = 12, L_x = 4, L_y = 4, L_z = -1;

    float *d_G, *h_G;
    size_t size = n * n * sizeof(float);

    /* initialize data on host */
    h_G = (float *) calloc(n * n, sizeof(float));
    assert(h_G);

    /* allocate device memory */
	hipError_t err = hipMalloc((void **) &d_G, size);
    hipMemset(d_G, 0, size); // Initialize the device memory to zero
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
	//cudaMemcpy(d_G, h_G, size, cudaMemcpyHostToDevice);

    hiprandState *d_states;

    hipError_t cudaStatus;
    /* Initialize cuRAND states */
    cudaStatus = hipMalloc((void **) &d_states, total_threads * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for cuRAND states: %s\n", hipGetErrorString(cudaStatus));
        return 1; // or exit(EXIT_FAILURE);
    }
    
    //printf("cudaMalloc status d_states: %s\n", cudaGetErrorString(cudaGetLastError()));
    initCurandStates<<<nblocks, ntpb>>>(d_states, total_threads);
    hipDeviceSynchronize(); // Ensure setup_states kernel finishes before proceeding
    //printf("synchronize status States: %s\n", cudaGetErrorString(cudaGetLastError())); 
    

    /* CUDA timers */
    hipEvent_t start_device, stop_device;  
    float elapsed_time;
    double time_device;

    /* creates CUDA timers but does not start yet */
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);


    //printf("launching kernel with %d blocks, %d threads per block\n", nblocks, ntpb);
    hipEventRecord( start_device, 0 );  

  	ray_tracing<<<nblocks,ntpb>>>(d_G, d_states, d_samples, N_rays, n, total_threads, L_x, L_y, L_z, C_y);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch ray_tracing kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //printf("kernel: %s\n", cudaGetErrorString(cudaGetLastError())); 
    
    
    hipEventRecord( stop_device, 0 );
    hipEventSynchronize( stop_device );
    hipEventElapsedTime( &elapsed_time, start_device, stop_device );
    time_device = static_cast<double>(elapsed_time); // Convert to double
    printf("kernel execution time: %f(s)\n",  time_device/1000.);
 

    /* copy data back to memory */
    hipMemcpy(h_G,d_G,n*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&samples,d_samples,sizeof(int), hipMemcpyDeviceToHost);

    //printf("cudaMemcpy status: %s\n", cudaGetErrorString(cudaGetLastError()));

    printf("num samples: %d\n",  samples);
    
    /* Save results to a file */
    char *filename = "double_sphere.bin";
    write_to_file(h_G, n, filename);

    free(h_G);
    hipFree(d_G);
    hipFree(d_samples);
    hipFree(d_states);
    hipEventDestroy( start_device );
    hipEventDestroy( stop_device );

    // Stop the clock and calculate the total execution time
    end_total = clock();
    double total_time = (double)(end_total - start_total) / CLOCKS_PER_SEC;
    printf("total execution time: %f(s)\n", total_time);

    return 0;

}
