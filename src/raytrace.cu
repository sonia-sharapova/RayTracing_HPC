#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

//#define ntpb 1024 // Number of threads per block
//#define nthreads_per_block 256 // Number of threads per block
//#define nthreads_per_block 128 // Number of threads per block
//#define W_y 2
//#define W_max 2
//#define R 6
//#define PI 3.14159265

__constant__ float W_y = 2;
__constant__ float W_max = 2;
__constant__ float R = 6;
__constant__ float PI = 3.14159265;

struct vector {
    float x, y, z;
};

__device__ long long atomicAdd64(long long* address, long long val) {
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        assumed + val);
    } while (assumed != old);
    //printf("samples:%lld\n",old);
    return old;
}



__global__ void initCurandStates(hiprandState *states, int numStates) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < numStates) {
        // Use id * 4238811 as the seed for each state for uniqueness
        hiprand_init(id * 4238811ULL, 0, 0, &states[id]);
    }
}

void write_to_file(float *G, int n, char *file_name) {
    FILE *fp = fopen(file_name, "wb"); // Open file in binary write mode
    if (fp == NULL) {
        perror("Error opening file");
        return;
    }

    for (int i = (n-1); i >= 0; i--) {
        // Write a row of n floats to the file as binary data
        if (fwrite(&G[i*n], sizeof(float), n, fp) != n) {
            perror("Error writing to file");
            break;
        }
    }
    fclose(fp);
}


__global__ void ray_tracing(float *G, hiprandState *states, int N_rays, int n, int total_threads, float L_x, float L_y, float L_z, float C_y) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localstate = states[id];
    //unsigned long long int s = 0;
    
    /* Predefined */
    vector W = {0.0f, W_y, 0.0f};
    vector C = {0.0f, C_y, 0.0f};
    vector L = {L_x, L_y, L_z};

    vector V,I,N,S;
        
    for(int ray = id; ray < N_rays; ray += total_threads){

        float temp = -1.0f;
        //int s = 0;
        //while (W.x > W_max || (W.x < -1.0*W_max) || W.z > W_max || (W.z < -1.0*W_max)|| temp <= 0.0) {
        while ((fabs(W.x) > W_max) || (fabs(W.z) > W_max) || temp <= 0.0f) {
            float phi = 1.0f * PI * hiprand_uniform(&localstate);
            float cosTheta = 2.0f * hiprand_uniform(&localstate) - 1.0f;
            float sinTheta = sqrt(1.0f - (cosTheta * cosTheta));

            V.x = sinTheta * (float) cos(phi);
            V.y = sinTheta * (float) sin(phi);
            V.z = cosTheta;

            //s+=1;
            //atomicAdd64(samples, 1LL);
            //atomicAdd(&samples, s);

            if (V.y != 0.0f) {
                W.x = (W.y / V.y) * V.x;
                W.z = (W.y / V.y) * V.z;

                float vc = (V.x * C.x) + (V.y * C.y) + (V.z * C.z);
                float cc = (C.x * C.x) + (C.y * C.y) + (C.z * C.z);
                temp = (vc * vc) + (R * R) - cc;
            }

            
        }

        //printf("W.x %f,W.z%f,temp%f\n",W.x,W.z,temp );
        //printf("W.x: %f, W.y:%f, W.z:%f\n", W.x, W.y, W.z);

        float t = (V.x * C.x) + (V.y * C.y) + (V.z * C.z) - sqrt(temp);
        I.x = t * V.x;
        I.y = t * V.y;
        I.z = t * V.z;

        N.x = (I.x - C.x) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));
        N.y = (I.y - C.y) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));
        N.z = (I.z - C.z) / sqrt((I.x - C.x)*(I.x - C.x) + (I.y - C.y)*(I.y - C.y) + (I.z - C.z)*(I.z - C.z));

        S.x = (L.x - I.x) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));
        S.y = (L.y - I.y) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));
        S.z = (L.z - I.z) / sqrt((L.x - I.x)*(L.x - I.x) + (L.y - I.y)*(L.y - I.y) + (L.z - I.z)*(L.z - I.z));

        float b = (S.x * N.x) + (S.y * N.y) + (S.z * N.z);
        if (b<0.0f)
            b = 0.0f;

        float normalizedWx = (W.x + W_max) / (2.0f * W_max);
        int i_index = (int)(normalizedWx * n);
        if (i_index < 0) i_index = 0;
        if (i_index >= n) i_index = n - 1;

        float normalizedWz = (W.z + W_max) / (2.0f * W_max);
        int j_index = (int)(normalizedWz * n);
        if (j_index < 0) j_index = 0;
        if (j_index >= n) j_index = n - 1;

        assert( (i_index < n) && (i_index>=0) );
        assert( (j_index < n) && (j_index>=0) );

        atomicAdd(&G[j_index + (n * i_index)], b);
        //*samples+=s;
        
    }
    // bug fix, set the state again to ensure continuity 
    states[id] = localstate;
    //atomicAdd(&samples, localDraws);
    //*samples+=s;
}


int main(int argc, char** argv) {

    if (argc != 5)
	{
		printf("4 arguments needed: number of rays, number of grid points, number of blocks, and threads per block. Exiting.\n");
		return 0;
	}

    clock_t start_total, end_total;
    start_total = clock();

    int N_rays = atoi(argv[1]);
    int n = atoi(argv[2]);
    int nblocks = atoi(argv[3]);
    int ntpb = atoi(argv[4]);

    /*
    unsigned long long int samples = 0;
    unsigned long long int *d_samples;

    cudaMalloc((void **)&d_samples, sizeof(unsigned long long int));
    cudaMemcpy(d_samples, &samples, sizeof(unsigned long long int), cudaMemcpyHostToDevice);
    */


    int total_threads = nblocks * ntpb;
    
    float C_y = 12.0f, L_x = 4.0f, L_y = 4.0f, L_z = -1.0f;

    float *d_G, *h_G;
    size_t size = n * n * sizeof(float);


    /* initialize data on host */
    h_G = (float *) calloc(n * n, sizeof(float));
    assert(h_G);

    /* allocate device memory */
	hipError_t err = hipMalloc((void **) &d_G, size);
    hipMemset(d_G, 0, size); // Initialize the device memory to zero
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
	//cudaMemcpy(d_G, h_G, size, cudaMemcpyHostToDevice);

    hiprandState *d_states;

    hipError_t cudaStatus;
    /* Initialize cuRAND states */
    cudaStatus = hipMalloc((void **) &d_states, total_threads * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for cuRAND states: %s\n", hipGetErrorString(cudaStatus));
        return 1; // or exit(EXIT_FAILURE);
    }
    
    //printf("cudaMalloc status d_states: %s\n", cudaGetErrorString(cudaGetLastError()));
    initCurandStates<<<nblocks, ntpb>>>(d_states, total_threads);
    hipDeviceSynchronize(); // Ensure setup_states kernel finishes before proceeding
    //printf("synchronize status States: %s\n", cudaGetErrorString(cudaGetLastError())); 
    

    /* CUDA timers */
    hipEvent_t start_device, stop_device;  
    float time_device;

    /* creates CUDA timers but does not start yet */
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);


    printf("launching kernel with %d blocks, %d threads per block\n", nblocks, ntpb);
    hipEventRecord( start_device, 0 );  

  	ray_tracing<<<nblocks,ntpb>>>(d_G, d_states, N_rays, n, total_threads, L_x, L_y, L_z, C_y);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch ray_tracing kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //printf("kernel: %s\n", cudaGetErrorString(cudaGetLastError())); 
    
    

    hipEventRecord( stop_device, 0 );
    hipEventSynchronize( stop_device );
    hipEventElapsedTime( &time_device, start_device, stop_device );
    printf("kernel execution time: %f(s)\n",  time_device/1000.);
 

    /* copy data back to memory */
    hipMemcpy(h_G,d_G,n*n*sizeof(float), hipMemcpyDeviceToHost);
    //cudaMemcpy(&samples,d_samples,sizeof(long long), cudaMemcpyDeviceToHost);
    printf("hipMemcpy status: %s\n", hipGetErrorString(hipGetLastError()));

    //printf("num samples: %lld\n",  samples);
    
    /* Save results to a file */
    char *filename = "cuda_sphere.bin";
    write_to_file(h_G, n, filename);

    free(h_G);
    hipFree(d_G);
    //cudaFree(d_samples);
    hipFree(d_states);
    hipEventDestroy( start_device );
    hipEventDestroy( stop_device );

    // Stop the clock and calculate the total execution time
    end_total = clock();
    float total_time = (float)(end_total - start_total) / CLOCKS_PER_SEC;
    printf("total execution time: %f(s)\n", total_time);

    return 0;

}
